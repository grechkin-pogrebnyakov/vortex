#include "hip/hip_runtime.h"
/*
 ============================================================================
 Name        : unit_1.cu
 Author      : Sergey Grechkin-Pogrebnyakov
 Version     : Mar. 02, 2015
 Copyright   : All rights reserved
 Description : unit_1 file of vortex project
 ============================================================================
 */

#include "unit_1.cuh"
#include "kernel.cuh"

TVars   *matr_creation(tPanel *panels, size_t s) {
    TVars rash = 0.0;
    size_t birth = 0;
    rash = (TVars)(s) / BLOCK_SIZE;
    birth = (size_t)(BLOCK_SIZE * ceil(rash));
    TVars **M = NULL;
    TVars *MM = NULL;
    TVars **L = NULL;
    L = new TVars*[s + 1];
    if (L == NULL) {
        return NULL;
    }
    {
        size_t i;
        for(i = 0; i < s+1; i++) {
            L[i] = NULL;
            L[i]=new TVars[s+1];
            if (L[i] == NULL) break;
        }
        if (i != s+1) {
            while (i != 0) {
                delete[] L[i--];
            }
            delete[] L;
        }
    }
    TVctr a_0 = {0.0, 0.0};
    TVctr a_1 = {0.0, 0.0};
    TVctr b_0 = {0.0, 0.0};
    TVctr b_1 = {0.0, 0.0};
    TVctr d_i = {0.0, 0.0};
    TVctr d_j = {0.0, 0.0};
    TVctr p_1, p_2, s_1, s_2;
    TVars q1_1, q1_2, q1_3, q2_1, q2_2, q2_3, z_1, z_2, z_3;
    TVctr c_1, c_2, c_3;
    TVctr tau, v;
    for (size_t i = 0; i < s; ++i) { 
        tau[0] = Tau_x( panels, i ); 
        tau[1] = Tau_y( panels, i );
        for (size_t j = 0;j < s; ++j) {
        if ( j != i ) {
            a_0[0] = R_left_x( panels, i ); 
            a_0[1] = R_left_y( panels, i );
            a_1[0] = R_right_x( panels, i ); 
            a_1[1] = R_right_y( panels, i );

            b_0[0] = R_left_x( panels, j ); 
            b_0[1] = R_left_y( panels, j );
            b_1[0] = R_right_x( panels, j ); 
            b_1[1] = R_right_y( panels, j );

            if ( ( j == i +1 ) || ( ( j == 0 ) && ( i == s - 1 ) ) ) {
                a_1[0] = R_left_x( panels, i ); 
                a_1[1] = R_left_y( panels, i );
                a_0[0] = R_right_x( panels, i ); 
                a_0[1] = R_right_y( panels, i );

                b_1[0] = R_left_x( panels, j ); 
                b_1[1] = R_left_y( panels, j );
                b_0[0] = R_right_x( panels, j ); 
                b_0[1] = R_right_y( panels, j );
            }

            d_j[0] = b_1[0] - b_0[0];
            d_j[1] = b_1[1] - b_0[1];
            d_i[0] = a_1[0] - a_0[0];
            d_i[1] = a_1[1] - a_0[1];
           
            p_1[0] = a_0[0] - b_1[0];
            p_1[1] = a_0[1] - b_1[1];
            p_2[0] = a_1[0] - b_1[0];
            p_2[1] = a_1[1] - b_1[1];

            s_1[0] = a_0[0] - b_0[0];
            s_1[1] = a_0[1] - b_0[1];
            s_2[0] = a_1[0] - b_0[0];
            s_2[1] = a_1[1] - b_0[1];

            z_1 = p_1[0] * p_2[1] - p_1[1] * p_2[0];
            z_2 = s_1[0] * s_2[1] - s_1[1] * s_2[0];
            z_3 = s_2[0] * p_2[1] - s_2[1] * p_2[0];

            if ( ( j == i - 1 ) || ( j == i + 1 )\
              || ( (j == 0 ) && ( i == s - 1 ) )\
              || ( (i == 0 ) && ( j == s - 1 ) ) ) {
                q1_1 = 0.0; q2_1 = 0.0;
            } else {
                q1_1 = atan( sp( d_i, p_1 ) / z_1 ) - atan( sp( d_i, p_2 ) / z_1 );
                q2_1 = 0.5 * log( sp( p_2, p_2 ) / sp( p_1, p_1 ) );
            }

            q1_2 = atan( sp( d_i, s_2 ) / z_2 ) - atan( sp( d_i, s_1 ) / z_2 );
            q1_3 = atan( sp( d_j, p_2 ) / z_3 ) - atan( sp( d_j, s_2 ) / z_3 );

            q2_2 = 0.5 * log( sp( s_1, s_1 ) / sp( s_2, s_2 ) );
            q2_3 = 0.5 * log( sp( p_2, p_2 ) / sp( s_2, s_2 ) );

            c_1[0] = sp( d_j, p_1 ) * d_i[0] + sp( d_i, s_1 ) * d_j[0] \
	 	   - sp( d_i, d_j ) * s_1[0];
            c_1[1] = sp( d_j, p_1 ) * d_i[1] + sp( d_i, s_1 ) * d_j[1] \
	 	   - sp( d_i, d_j ) * s_1[1];
            c_2[0] = c_1[0] + sp( d_j, d_j ) * d_i[0];
            c_2[1] = c_1[1] + sp( d_j, d_j ) * d_i[1];
            c_3[0] = sp( d_i, d_i ) * d_j[0];
            c_3[1] = sp( d_i, d_i ) * d_j[1];

            v[0] = 1.0 / ( 2 * M_PI * sqrt( sp( d_j, d_j ) ) * sp( d_i, d_i ) )\
                 * (q1_1 * c_1[0] + q1_2 * c_2[0] + q1_3 * c_3[0]\
                  + ( q2_1 * c_1[1] + q2_2 * c_2[1] + q2_3 * c_3[1] ) );
            v[1] = 1.0 / ( 2 * M_PI * sqrt( sp( d_j, d_j ) ) * sp( d_i, d_i ) )\
                 * (q1_1 * c_1[1] + q1_2 * c_2[1] + q1_3 * c_3[1]\
                  - ( q2_1 * c_1[0] + q2_2 * c_2[0] + q2_3 * c_3[0] ) );

            L[i][j] = sp( v, tau );
        } else L[i][j] = -0.5;
        }
    }
    for (size_t i=0; i<s; i++) {
        L[s][i] = Panel_length( panels, i );
        L[i][s]=1;
    }
    L[s][s]=0;
    save_matr(L, s+1, "L.txt");
    M=inverse_matrix(L,s+1);
	clear_memory(L, s+1);
    if (M == NULL) {    
        return NULL;
    }
	save_matr(M, s+1, "M.txt");
	MM = new TVars[(birth + 1) * (birth + 1)];
    if (MM == NULL) {
		clear_memory(M, s+1);
		return NULL;
	}
    for (size_t i=0; i < s+1; i++) {
        for (size_t j=0; j < s+1 ; j++) {
            MM[(birth+1)*i+j]=M[i][j];
        }
        for (size_t j=(s+1); j<(birth+1);j++) {
            MM[(birth+1)*i+j]=0;
        }
    }
    for (size_t i=s+1; i < birth+1; i++) {
        for (size_t j=0; j<(birth+1);j++) {
            MM[(birth+1)*i+j]=0;
        }
    }
    clear_memory(M, s+1);
    return MM;
}

TVars   *load_matrix(size_t &p) {
	using namespace std;
	ifstream infile;
	infile.open("M.txt");
	infile >> p;
	p--;
	TVars **M = NULL;
	M = new TVars*[p + 1];
    if (M == NULL) {
        return NULL;
    }
	{
        size_t i;
        for(i = 0; i < p+1; i++) {
            M[i] = NULL;
            M[i]=new TVars[p+1];
            if (M[i] == NULL) break;
        }
        if (i != p+1) {
            while (i != 0) {
                delete[] M[i--];
            }
            delete[] M;
        }
    }
	for (size_t i = 0; i < p + 1; ++i) {
		for (size_t j = 0; j < p + 1; ++j) {
			infile >> M[i][j];
		}
	}
    TVars rash = 0.0;
    size_t birth = 0;
    rash = (TVars)(p) / BLOCK_SIZE;
    birth = (size_t)(BLOCK_SIZE * ceil(rash));
    
    TVars *MM = NULL;
    MM = new TVars[(birth + 1) * (birth + 1)];
    if (MM == NULL) return NULL;

    for (size_t i=0; i < p+1; i++) {
        for (size_t j=0; j < p+1 ; j++) {
            MM[(birth+1)*i+j]=M[i][j];
        }
        for (size_t j=(p+1); j<(birth+1);j++) {
            MM[(birth+1)*i+j]=0;
        }
    }
    for (size_t i=p+1; i < birth+1; i++) {
        for (size_t j=0; j<(birth+1);j++) {
            MM[(birth+1)*i+j]=0;
        }
    }
    clear_memory(M, p+1);
    return MM;
}

int     save_matr(TVars* M, size_t size, char *name = "D.txt") {
    using namespace std;
    if (M == NULL) return 1;
    ofstream outfile;
    outfile.open(name);
    for (size_t i = 0; i < size; ++i) {
        for (size_t j = 0; j < size; j++) {
            outfile<<(TVars)(M[i*size+j])<<"    ";
        }// for j
        outfile<<endl;
    }// for i
    outfile.close();
    return 0;
}
int     save_matr(TVars** M, size_t size, char *name = "D.txt") {
    using namespace std;
    if (M == NULL) return 1;
    ofstream outfile;
    outfile.open(name);
	outfile << size << '\n';
    for (size_t i = 0; i < size; ++i) {
        if (M[i] == NULL) {
            outfile.close();
            return 1;
        }
        for (size_t j = 0; j < size; j++) {
            outfile<<(TVars)(M[i][j])<<"    ";
        }// for j
        outfile<<endl;
    }// for i
    outfile.close();
    return 0;
}
TVars   **inverse_matrix(TVars **M, size_t size) {
    int err = 0;
    size_t *POR = NULL;                                        // массив для учёта перестановки строк
    POR = new size_t[size];
    if (!POR) return NULL;
    size_t PR;                                                 // переменная для учёта перестановок строк
    for (size_t i = 0; i < size; i++) {
        POR[i]=i;
    }
    TVars b;
    TVars **M_inv = NULL;                                   // обратная матрица
    M_inv = new TVars*[size];
    {
        size_t i;
        for(i = 0; i < size; ++i) {
            M_inv[i] = NULL;
            M_inv[i]=new TVars[size];
            if (!M_inv[i]) break;
        }
        if (i != size) {
            while (i != 0) {
                delete[] M_inv[i--];
            }
            delete [] M_inv;
            delete[] POR;
            return NULL;
        }
    }
    for (size_t i = 0; i < size; i++) {
        for (size_t j = 0; j < size; j++) {
            if (i != j) {
                M_inv[i][j]=0;
            }
            else {
                M_inv[i][j]=1;
            }
        }
    }
    if (fabs(M[0][0])<DELT) {
        TVars mx=fabs(M[0][0]);
        size_t num=0;
        for (size_t i = 1; i < size; i++) {
            if (fabs(M[i][0])>mx) {
                mx=fabs(M[i][0]);
                num=i;
            }
        }//i
        if (num!=0) {
            err = move_line(M,size,0,num);
            if (err) {
                move_all_back(M, size, POR);
                delete[] POR;
                clear_memory(M_inv, size);
                return NULL;
            }
            err = move_line(M_inv,size,0,num);
            PR=POR[0];
            POR[0]=POR[num];
            POR[num]=PR;
            if (err) {
                move_all_back(M, size, POR);
                delete[] POR;
                clear_memory(M_inv, size);
                return NULL;
            }
        }
    }//if
    for (size_t k = 0; k < size-1; k++) {
        if (fabs(M[k][k])<DELT) {
            move_all_back(M, size, POR);
            delete[] POR;
            clear_memory(M_inv, size);
            return NULL;
        }//if
        TVars mx=fabs(M[k+1][k+1]);
        size_t line=k+1;
        for (size_t i = k+1; i < size; i++) {               // Выбор главного элемента
            if (fabs(M[i][k+1])>mx) {
                mx=fabs(M[i][k+1]);
                line=i;
            }//if
        }//i
        if (mx<DELT) {
            move_all_back(M, size, POR);
            delete[] POR;
            clear_memory(M_inv, size);
            return NULL;
        }
        err = move_line(M,size,k+1,line);
        if (err) {
            move_all_back(M, size, POR);
            delete[] POR;
            clear_memory(M_inv, size);
            return NULL;
        }
        err = move_line(M_inv,size,k+1,line);                      // перестановка строк
        PR=POR[k+1];
        POR[k+1]=POR[line];
        POR[line]=PR;
        if (err) {
            move_all_back(M, size, POR);
            delete[] POR;
            clear_memory(M_inv, size);
            return NULL;
        }
        for (size_t i = 0; i < size; i++) {
            if (i!=k) {
                TVars c=M[i][k]/M[k][k];
                for (size_t j = 0; j < size; j++) {
                    b=M[i][j]-c*(M[k][j]);                  // преобразование матрицы
                    M[i][j]=b;
                    b=M_inv[i][j]-c*(M_inv[k][j]);          // преобразование матрицы
                    M_inv[i][j]=b;
                }//j
            }//if
        }//i
    }//k
    if (fabs(M[size-1][size-1])<DELT) {
        move_all_back(M, size, POR);
        delete[] POR;
        clear_memory(M_inv, size);
        return NULL;
    }
    for (size_t i = 0; i < size-1; ++i) {
        TVars c=M[i][size-1]/M[size-1][size-1];
    //		   b=M[i][size-1]-c*(M[size-1][size-1]);        // преобразование матрицы
    //		   M[i][size-1]=b;
        for (size_t j = 0; j < size; j++) {
            b=M_inv[i][j]-c*(M_inv[size-1][j]);                 // преобразование матрицы
            M_inv[i][j]=b;
        }// j
    }// i
    for (size_t i = 0; i < size; i++) {
        for (size_t j = 0; j < size; j++) {
            M_inv[i][j]=M_inv[i][j]/M[i][i];
        }// j
    }// i
    move_all_back(M, size, POR);
    delete[] POR;
    return M_inv;
}
int     move_line(TVars **M, size_t s, size_t st, size_t fin) {
    TVars *Ln = NULL;
    Ln=new TVars[s];
    if (! Ln) return 1;
    for (size_t i = 0; i < s ; i++) {
        Ln[i]=M[st][i];
    }
    for (size_t i = 0; i < s ; i++) {
        M[st][i]=M[fin][i];
    }
    for (size_t i = 0; i < s ; i++) {
        M[fin][i]=Ln[i];
    }
    delete[] Ln;
    return 0;
}
int     move_all_back(TVars **M, size_t size, size_t *mov) {
    if (M == NULL || mov == NULL) return 1;
    int err = 0;
    int cnt = 0;
    for (size_t i = 0; i < size; ++i) {
        if (mov[i] != i) {
            err = move_line(M, size, i, mov[i]);
            if (err || cnt < 10) {
                --i;
                ++cnt;
                continue;
            }
            cnt = 0;
            if (err) return 1;
            mov[mov[i]] = mov[i];
            mov[i] = i;
        }
    }
    return 0;
}
void    clear_memory(TVars **M, size_t s) {
    if (M != NULL) {
        for (size_t i = 0; i < s; ++i) {
            if (M[i] != NULL) {
                delete[] M[i];
            }
        }
        delete[] M;
    }
}
int     incr_vort_quont(Vortex *&p_host, Vortex *&p_dev, PVortex *&v_host, PVortex *&v_dev, TVars *&d_dev, size_t &size)
{
    using namespace std;
    hipError_t cuerr;
    if (p_host != NULL && p_dev != NULL && v_host != NULL && v_dev != NULL && d_dev != NULL)
    {
        Vortex *p_dev_new = NULL;
        size_t size_n = size + INCR_STEP;
        cuerr=hipMalloc( (void**)&p_dev_new , size_n * sizeof(Vortex));
        if (cuerr != hipSuccess) {
            cout << hipGetErrorString(cuerr) << '\n';
            return 1;
        }
        cuerr = hipMemcpy (p_dev_new, p_dev, size  * sizeof(Vortex), hipMemcpyDeviceToDevice);
        if (cuerr != hipSuccess) {
            cout << hipGetErrorString(cuerr) << '\n';
            return 1;
        }
        size += INCR_STEP;
        delete[] p_host;
        p_host = new Vortex[size];
        delete[] v_host;
        v_host = new PVortex[size];
        hipFree(p_dev);
        hipFree(d_dev);
        hipFree(v_dev);
        cuerr=hipMalloc( (void**)&d_dev, size * sizeof(TVars));
        if (cuerr != hipSuccess) {
            cout << hipGetErrorString(cuerr) << '\n';
            return 1;
        }
        cuerr=hipMalloc( (void**)&v_dev, size  * sizeof(PVortex));
        if (cuerr != hipSuccess) {
            cout << hipGetErrorString(cuerr) << '\n';
            return 1;
        }
        p_dev = p_dev_new;
        hipDeviceSynchronize();
    }
    else {
        size = INCR_STEP;
        p_host=new Vortex[size];
        v_host=new PVortex[size];
        cuerr=hipMalloc((void**)&p_dev , size * sizeof(Vortex));
        if (cuerr != hipSuccess) {
            cout << hipGetErrorString(cuerr) << '\n';
            return 1;
        }
        cuerr=hipMalloc((void**)&d_dev , size * sizeof(TVars));
        if (cuerr != hipSuccess) {
            cout << hipGetErrorString(cuerr) << '\n';
            return 1;
        }
        cuerr=hipMalloc((void**)&v_dev , size  * sizeof(PVortex));
        if (cuerr != hipSuccess) {
            cout << hipGetErrorString(cuerr) << '\n';
            return 1;
        }
    }
    srand((unsigned int)time(NULL));
    float *rnd_dev = NULL, *rnd_host = NULL;
    rnd_host = new float[INCR_STEP];
    for (int i = 0; i < INCR_STEP; ++i) {
        rnd_host[i] = (float)rand();
    }
    cuerr = hipMalloc((void**)&rnd_dev, INCR_STEP * sizeof(float));
    if (cuerr != hipSuccess) {
        cout << hipGetErrorString(cuerr) << '\n';
        return 1;
    }
    cuerr = hipMemcpy(rnd_dev, rnd_host, INCR_STEP * sizeof(float), hipMemcpyHostToDevice);
    if (cuerr != hipSuccess) {
        cout << hipGetErrorString(cuerr) << '\n';
        return 1;
    }
    dim3 threads = dim3(BLOCK_SIZE);
    dim3 blocks  = dim3(INCR_STEP/BLOCK_SIZE);
    // generate random numbers
    zero_Kernel <<< blocks, threads >>> (rnd_dev, p_dev, (size-INCR_STEP) );
    hipDeviceSynchronize();
    //	cuerr=hipMemcpy ( p_host , p_dev , size  * sizeof(Vortex) , hipMemcpyDeviceToHost);
    //	save_to_file_size(1);
    cuerr = hipGetLastError();
    if (cuerr != hipSuccess) {
        cout << hipGetErrorString(cuerr) << '\n';
        return 1;
    }
    return 0;
}
int     vort_creation(Vortex *pos, TVctr *V_infDev, size_t n_of_birth, size_t n_of_birth_BLOCK_S,
                     size_t n, TVars * M_Dev, TVars *d_g, tPanel *panels) {
    using namespace std;
    hipError_t cuerr = hipSuccess;
    hipEvent_t start, stop;
start_timer(start, stop);
    TVars *R_p = NULL;
    cuerr=hipMalloc((void**)&R_p, (n_of_birth_BLOCK_S) * sizeof(TVars));
    if (cuerr != hipSuccess) {
        cout << hipGetErrorString(cuerr) << '\n';
        return 1;
    }
    extern int current_step;

	dim3 threads1 = dim3(BLOCK_SIZE);
    dim3 blocks1  = dim3(n_of_birth_BLOCK_S/BLOCK_SIZE);
    Right_part_Kernel <<< blocks1, threads1 >>> (pos, V_infDev, n, n_of_birth_BLOCK_S, R_p, panels);
	hipDeviceSynchronize();
    cuerr = hipGetLastError();
    if (cuerr != hipSuccess) {
        cout << hipGetErrorString(cuerr) << '\n';
        return 1;
    }


	birth_Kernel<<< blocks1, threads1 >>>(pos, n, n_of_birth, n_of_birth_BLOCK_S, M_Dev, d_g, R_p, panels);
	hipDeviceSynchronize();
    cuerr = hipGetLastError();
    if (cuerr != hipSuccess) {
        cout << hipGetErrorString(cuerr) << '\n';
        return 1;
    }
    hipFree(R_p);
    return 0;
}
void start_timer(hipEvent_t &start, hipEvent_t &stop) {
    hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start,0);
	hipEventSynchronize(start);
}
float stop_timer(hipEvent_t start, hipEvent_t stop) {
    hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	float time;
	hipEventElapsedTime(&time,start,stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	time=time/1000;
    return time;
}
int Speed(Vortex *pos, TVctr *v_inf, size_t s, PVortex *v, TVars *d, TVars nu, tPanel *panels) {
    //extern int current_step;
    //extern size_t n;
    hipError_t cuerr = hipSuccess;
	hipDeviceSynchronize();
	dim3 threads = dim3(BLOCK_SIZE);
    dim3 blocks  = dim3(s/BLOCK_SIZE);
    //PVortex * VEL = new PVortex[s];
    //PVortex * VELLL = new PVortex[s];
	shared_Kernel <<< blocks, threads >>> (pos, v_inf, s, v, d);
//	simple_Kernel <<< blocks, threads >>> (pos, v_inf, *n, v);
    hipDeviceSynchronize();
    //Vortex *POS = new Vortex[s];
    //cuerr=hipMemcpy (POS  , pos , s  * sizeof(Vortex) , hipMemcpyDeviceToHost);
    //cuerr=hipMemcpy (VEL  , v , s  * sizeof(PVortex) , hipMemcpyDeviceToHost);
    //save_vel_to_file(POS, VEL, n, current_step, 0);
    cuerr=hipGetLastError(); 
	if (cuerr != hipSuccess) {               
		std::cout <<hipGetErrorString(cuerr);
		return 1;
	}//if

//	TVars* dd=new TVars[s];
//    hipMemcpy(dd,d,s * sizeof(TVars),hipMemcpyDeviceToHost);
//    save_d(dd, s, current_step);
//    delete[]dd;

	diffusion_Kernel <<< blocks, threads >>> (pos, s, v, d, nu);
//	cuerr=hipMemcpy (POS  , posDev , size  * sizeof(Vortex) , hipMemcpyDeviceToHost);
//	save_to_file(j);
	hipDeviceSynchronize();
//	cuerr=hipMemcpy (VELLL  , v , s  * sizeof(PVortex) , hipMemcpyDeviceToHost);
/*
    for (size_t sss = 0; sss < s; ++sss) {
        VEL[sss].v[0] = VELLL[sss].v[0] - VEL[sss].v[0];
        VEL[sss].v[1] = VELLL[sss].v[1] - VEL[sss].v[1];
    }
	save_vel_to_file(POS, VEL, n, current_step, 1);
*/
    cuerr=hipGetLastError(); 
	if (cuerr != hipSuccess) {               
		std::cout <<hipGetErrorString(cuerr);
		return 1;
	}//if
	diffusion_2_Kernel <<< blocks, threads >>> (pos, s, v, d, nu, panels);
//	cuerr=hipMemcpy (VEL  , VDev , size  * sizeof(PVortex) , hipMemcpyDeviceToHost);
//	stf(j,1);
	hipDeviceSynchronize();
/*
    cuerr=hipMemcpy (VEL  , v , s  * sizeof(PVortex) , hipMemcpyDeviceToHost);
    for (size_t sss = 0; sss < s; ++sss) {
        VELLL[sss].v[0] = VEL[sss].v[0] - VELLL[sss].v[0];
        VELLL[sss].v[1] = VEL[sss].v[1] - VELLL[sss].v[1];
    }
    save_vel_to_file(POS, VELLL, n, current_step, 2);
    save_vel_to_file(POS, VEL, n, current_step, 3);
*/
/*	
	TVars *dd=new TVars[size];
    hipMemcpy(dd,d,size * sizeof(TVars),hipMemcpyDeviceToHost);
    cout<<"d= "<<dd[0]<<endl;
    delete[]dd;
	TVars *ddt=new TVars;
	cuerr=hipMemcpy (ddt  , den ,  sizeof(TVars) , hipMemcpyDeviceToHost);
	cout<<"nu*y=  "<<(*ddt)<<endl;
	TVctr V_inf;
	cuerr=hipMemcpy (VEL  , VDev , size  * sizeof(PVortex) , hipMemcpyDeviceToHost);
	cuerr=hipMemcpy (V_inf  , V_infDev , sizeof(TVctr) , hipMemcpyDeviceToHost);
	cout<<"V=  "<<VEL[0].v[0]<<endl;
*/
//	hipDeviceSynchronize();
//    cuerr=hipMemcpy2D ( M3 , nb , cDev, pitch , nb , &n , hipMemcpyDeviceToHost);
   	cuerr=hipGetLastError();
	if (cuerr != hipSuccess) {
		std::cout <<hipGetErrorString(cuerr);
		return 1;
	}//if
	return 0;
}

void save_vel_to_file(Vortex *POS, PVortex *VEL, size_t size, int _step, int stage) {
    using namespace std;
    char *fname1;
    fname1 = "velocities/Vel";
    char *fname2;
    fname2 = ".txt";
    char *fzero;
    fzero = "0";
    char fstep[8];
    char fname[20];
    fname[0] = '\0';
    char stage_str[5];
    sprintf(stage_str, "%d", stage);
    sprintf(fstep,"%d", _step);
    strcat(fname,fname1);
    strcat(fname, stage_str);
    if (_step<10) strcat(fname,fzero);
    if (_step<100) strcat(fname,fzero);
    if (_step<1000) strcat(fname,fzero);
    if (_step<10000) strcat(fname,fzero);
    //	if (_step<100000) strcat(fname,fzero);
    strcat(fname,fstep);
    strcat(fname,fname2);
    ofstream outfile;
    outfile.open(fname);
    // ���������� ����� ������ � ������
    outfile << (size) << endl;
    for (size_t i = 0; i < (size); ++i) {
        outfile<<(int)(i)<<" "<<(TVars)(POS[i].r[0])<<" "<<(TVars)(POS[i].r[1])<<" "<<(TVars)(VEL[i].v[0])<<" "<<(TVars)(VEL[i].v[1])<<endl;
        //      outfile<<(TVars)(d[i])<<" "<<(TVars)(POS[i].r[0])<<" "<<(TVars)(POS[i].r[1])<<" "<<(TVars)(POS[i].g)<<endl;     
    }//for i
    outfile.close();
} //save_to_file

void save_d(TVars *d, size_t size, int _step) {
    using namespace std;
    char *fname1;
    fname1 = "ddd/d";
    char *fname2;
    fname2 = ".txt";
    char *fzero;
    fzero = "0";
    char fstep[8];
    char fname[20];
    fname[0] = '\0';
    sprintf(fstep,"%d", _step);
    strcat(fname,fname1);
    if (_step<10) strcat(fname,fzero);
    if (_step<100) strcat(fname,fzero);
    if (_step<1000) strcat(fname,fzero);
    if (_step<10000) strcat(fname,fzero);
    //	if (_step<100000) strcat(fname,fzero);
    strcat(fname,fstep);
    strcat(fname,fname2);
    ofstream outfile;
    outfile.open(fname);
    // ���������� ����� ������ � ������
    outfile << (size) << endl;
    for (size_t i = 0; i < (size); ++i) {
        outfile<<(int)(i)<<" "<<d[i]<<endl;
        //      outfile<<(TVars)(d[i])<<" "<<(TVars)(POS[i].r[0])<<" "<<(TVars)(POS[i].r[1])<<" "<<(TVars)(POS[i].g)<<endl;     
    }//for i
    outfile.close();
} //save_to_file

int Step(Vortex *pos, PVortex *V, size_t &n, size_t s, TVars *d_g, PVortex *F_p, TVars *M, tPanel *panels) {
	hipError_t cuerr = hipSuccess;
	TVars *d_g_Dev = NULL;
	cuerr=hipMalloc((void**)&d_g_Dev, n * sizeof(TVars));
	if (cuerr!= hipSuccess) {
		std::cout <<hipGetErrorString(cuerr) << '\n';
		return 1;
	}//if
    PVortex *F_p_dev = NULL;
    TVars *M_dev = NULL;
    cuerr=hipMalloc((void**)&F_p_dev, n * sizeof(PVortex));
	if (cuerr!= hipSuccess) {
		std::cout <<hipGetErrorString(cuerr) << '\n';
		return 1;
	}//if
    cuerr=hipMalloc((void**)&M_dev, n * sizeof(TVars));
	if (cuerr!= hipSuccess) {
		std::cout <<hipGetErrorString(cuerr) << '\n';
		return 1;
	}//if
//	TVars d_g_h;
//	cuerr=hipMemcpy ( &d_g_h, d_g , sizeof(TVars) , hipMemcpyDeviceToHost);
//  std::cout << "D_g_before = " << d_g_h << '\n';
    dim3 threads = dim3(BLOCK_SIZE);
    dim3 blocks  = dim3(s/BLOCK_SIZE);
	step_Kernel <<< blocks, threads >>> (pos, V, d_g_Dev, F_p_dev, M_dev, n, panels);
    hipDeviceSynchronize();
    cuerr=hipGetLastError();
	if (cuerr!= hipSuccess) {
		std::cout <<hipGetErrorString(cuerr) << '\n';
		return 1;
	}//if

//	cuerr=hipMemcpy ( POS , posDev , size  * sizeof(Vortex) , hipMemcpyDeviceToHost);
//	save_to_file_size((*n)+1);

	summ_Kernel <<< dim3(1),dim3(1) >>> (d_g_Dev, d_g, F_p_dev, F_p, M_dev, M, n);
	hipDeviceSynchronize();
    cuerr=hipGetLastError();
	if (cuerr!= hipSuccess) {
		std::cout <<hipGetErrorString(cuerr) << '\n';
		return 1;
	}//if
	hipFree(d_g_Dev);
    hipFree(F_p_dev);
    hipFree(M_dev);
	TVars d_g_h = 0.0;
	cuerr=hipMemcpy ( &d_g_h, d_g , sizeof(TVars) , hipMemcpyDeviceToHost);
//	std::cout << "d_g = " << d_g_h << '\n';

	size_t *n_dev = NULL;
	cuerr = hipMalloc( (void**)&n_dev ,  sizeof(size_t)); 
	if (cuerr!= hipSuccess) {               
		std::cout <<hipGetErrorString(cuerr) << '\n';
		return 1;
	}//if
	cuerr = hipMemcpy(n_dev, &n, sizeof(size_t), hipMemcpyHostToDevice);
	if (cuerr!= hipSuccess) {
		std::cout <<hipGetErrorString(cuerr) << '\n';
		return 1;
	}//if
	sort_Kernel <<< dim3(1), dim3(1) >>> (pos,n_dev);
    hipDeviceSynchronize();
    cuerr=hipGetLastError();
	if (cuerr!= hipSuccess) {
		std::cout <<hipGetErrorString(cuerr) << '\n';
		return 1;
	}//if
//	std::cout << "n_old =  " << n;
	cuerr = hipMemcpy(&n,n_dev,sizeof(size_t), hipMemcpyDeviceToHost);
	if (cuerr!= hipSuccess) {
		std::cout <<hipGetErrorString(cuerr) << '\n';
		return 1;
	}//if
//        std::cout << "   n_new =  " << n <<'\n';
	hipFree(n_dev);
//    std::cout << "first collapse\n";
	for (int cc = 0; cc < NCOL; ++cc) {
		int *Setx = NULL;
		int *Sety = NULL;
		int *COL = NULL;
		cuerr=hipMalloc (&Setx, n * sizeof( int ));
		cuerr=hipMalloc (&Sety, n * sizeof( int ));
		cuerr=hipMalloc (&COL, n * sizeof( int ));
		
		first_setka_Kernel <<< blocks, threads >>> (pos, n, Setx, Sety, COL);
		hipFree(Setx);
		hipFree(Sety);
//		int *COLD;
//		COLD= new int [n];
//		hipMemcpy(COLD, COL, n * sizeof(int), hipMemcpyDeviceToHost);
/*
        int sss = 0;
		for(int gg = 0; gg < n; gg++) {
			if (COLD[gg] >= 0) sss += 1;
		}
		std::cout << cc << ' ' << sss << '\n';
		if (sss==0) cc=10;
		delete[] COLD;
*/
        hipDeviceSynchronize();
		first_collapse_Kernel <<< dim3(1), dim3(1) >>> (pos, COL, n);
		hipFree(COL);
		hipMalloc( (void**)&n_dev ,  sizeof(size_t));
		hipMemcpy(n_dev, &n, sizeof(size_t), hipMemcpyHostToDevice);
		hipDeviceSynchronize();
		sort_Kernel <<< dim3(1), dim3(1) >>> (pos, n_dev);
		hipDeviceSynchronize();
		hipMemcpy(&n, n_dev, sizeof(size_t), hipMemcpyDeviceToHost);
		hipFree(n_dev);
	}
//    std::cout << "second collapse\n";
    for (int cc = 0; cc < NCOL; ++cc) {
        int *Setx = NULL;
        int *Sety = NULL;
        int *COL = NULL;
        cuerr=hipMalloc (&Setx, n * sizeof( int ));
        cuerr=hipMalloc (&Sety, n * sizeof( int ));
        cuerr=hipMalloc (&COL, n * sizeof( int ));

        second_setka_Kernel <<< blocks, threads >>> (pos, n, Setx, Sety, COL);
        hipFree(Setx);
        hipFree(Sety);
/*
        int *COLD;
        COLD= new int [n];
        hipMemcpy(COLD, COL, n * sizeof(int), hipMemcpyDeviceToHost);
        int sss = 0;
        for(int gg = 0; gg < n; gg++) {
            if (COLD[gg] >= 0) sss += 1;
        }
        std::cout << cc << ' ' << sss << '\n';
        if (sss==0) cc=10;
        delete[] COLD;
*/
        hipDeviceSynchronize();
        second_collapse_Kernel <<< dim3(1), dim3(1) >>> (pos, COL, n);
        hipFree(COL);
        hipMalloc( (void**)&n_dev ,  sizeof(size_t));
        hipMemcpy(n_dev, &n, sizeof(size_t), hipMemcpyHostToDevice);
        hipDeviceSynchronize();
        sort_Kernel <<< dim3(1), dim3(1) >>> (pos, n_dev);
        hipDeviceSynchronize();
        hipMemcpy(&n, n_dev, sizeof(size_t), hipMemcpyDeviceToHost);
        hipFree(n_dev);
    }
	return 0;
}



int velocity_control(Vortex *pos, TVctr *V_inf, int n, PVortex *Contr_points, PVortex *V, int *n_v) {
    hipError_t cuerr;
    size_t nummm = 500;
    TVars rash = 0.0;
    size_t birth = 0;
    rash = (TVars)(nummm) / BLOCK_SIZE;
    birth = (size_t)(BLOCK_SIZE * ceil(rash));
    dim3 threads = dim3(BLOCK_SIZE);
    dim3 blocks  = dim3(birth / BLOCK_SIZE);
    velocity_control_Kernel <<< blocks, threads >>> (pos, V_inf, n, Contr_points, V, n_v);
    hipDeviceSynchronize();    
    cuerr = hipGetLastError();    
    if (cuerr != hipSuccess) {        
        std::cout << hipGetErrorString(cuerr) << '\n';
        return 1;
    }
    return 0;
}
