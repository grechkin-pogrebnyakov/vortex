#include "hip/hip_runtime.h"
#include "unit_1.cuh"

TVars   *matr_creation(tPanel *panels, size_t s) {
    double rash = 0.0;
    size_t birth = 0;
    rash = (double)(s) / BLOCK_SIZE;
    birth = (size_t)(BLOCK_SIZE * ceil(rash));
    TVars **M = NULL;
    TVars *MM = NULL;
    TVars **L = NULL;
    L = new TVars*[s + 1];
    if (L == NULL) {
        return NULL;
    }
    {
        size_t i;
        for(i = 0; i < s+1; i++) {
            L[i] = NULL;
            L[i]=new TVars[s+1];
            if (L[i] == NULL) break;
        }
        if (i != s+1) {
            while (i != 0) {
                delete[] L[i--];
            }
            delete[] L;
        }
    }
    double dist2 = 0.0;
    TVctr a = {0.0, 0.0};
    TVctr N = {0.0, 0.0};
    TVctr b = {0.0, 0.0};
    for (size_t i = 0; i < s; ++i) { 
        a[0] = R_contr_x(panels, i);
        a[1] = R_contr_y(panels, i);
        N[0] = N_contr_x(panels, i);
        N[1] = N_contr_y(panels, i);
        for (size_t j = 0;j < s; ++j) {
            b[0] = R_birth_x(panels, j);
            b[1] = R_birth_y(panels, j);
            dist2 = Ro2(a, b);
		    dist2 = max(dist2,EPS2);
            L[i][j] = ((a[0] - b[0]) * N[1] -
            (a[1] - b[1]) * N[0]) /
            (2 * M_PI * dist2);
        }
    }
    for (size_t i=0; i<s; i++) {
        L[s][i]=1;
        L[i][s]=1;
    }
    L[s][s]=0;
    save_matr(L, s+1, "L.txt");
    M=inverse_matrix(L,s+1);
	clear_memory(L, s+1);
    if (M == NULL) {    
        return NULL;
    }
	save_matr(M, s+1, "M.txt");
	MM = new TVars[(birth + 1) * (birth + 1)];
    if (MM == NULL) {
		clear_memory(M, s+1);
		return NULL;
	}
    for (size_t i=0; i < s+1; i++) {
        for (size_t j=0; j < s+1 ; j++) {
            MM[(birth+1)*i+j]=M[i][j];
        }
        for (size_t j=(s+1); j<(birth+1);j++) {
            MM[(birth+1)*i+j]=0;
        }
    }
    for (size_t i=s+1; i < birth+1; i++) {
        for (size_t j=0; j<(birth+1);j++) {
            MM[(birth+1)*i+j]=0;
        }
    }
    clear_memory(M, s+1);
    return MM;
}

TVars   *load_matrix(size_t &p) {
	using namespace std;
	ifstream infile;
	infile.open("M.txt");
	infile >> p;
	p--;
	TVars **M = NULL;
	M = new TVars*[p + 1];
    if (M == NULL) {
        return NULL;
    }
	{
        size_t i;
        for(i = 0; i < p+1; i++) {
            M[i] = NULL;
            M[i]=new TVars[p+1];
            if (M[i] == NULL) break;
        }
        if (i != p+1) {
            while (i != 0) {
                delete[] M[i--];
            }
            delete[] M;
        }
    }
	for (size_t i = 0; i < p + 1; ++i) {
		for (size_t j = 0; j < p + 1; ++j) {
			infile >> M[i][j];
		}
	}
    double rash = 0.0;
    size_t birth = 0;
    rash = (double)(p) / BLOCK_SIZE;
    birth = (size_t)(BLOCK_SIZE * ceil(rash));
    
    TVars *MM = NULL;
    MM = new TVars[(birth + 1) * (birth + 1)];
    if (MM == NULL) return NULL;

    for (size_t i=0; i < p+1; i++) {
        for (size_t j=0; j < p+1 ; j++) {
            MM[(birth+1)*i+j]=M[i][j];
        }
        for (size_t j=(p+1); j<(birth+1);j++) {
            MM[(birth+1)*i+j]=0;
        }
    }
    for (size_t i=p+1; i < birth+1; i++) {
        for (size_t j=0; j<(birth+1);j++) {
            MM[(birth+1)*i+j]=0;
        }
    }
    clear_memory(M, p+1);
    return MM;
}

int     save_matr(TVars* M, size_t size, char *name = "D.txt") {
    using namespace std;
    if (M == NULL) return 1;
    ofstream outfile;
    outfile.open(name);
    for (size_t i = 0; i < size; ++i) {
        for (size_t j = 0; j < size; j++) {
            outfile<<(double)(M[i*size+j])<<"    ";
        }// for j
        outfile<<endl;
    }// for i
    outfile.close();
    return 0;
}
int     save_matr(TVars** M, size_t size, char *name = "D.txt") {
    using namespace std;
    if (M == NULL) return 1;
    ofstream outfile;
    outfile.open(name);
	outfile << size << '\n';
    for (size_t i = 0; i < size; ++i) {
        if (M[i] == NULL) {
            outfile.close();
            return 1;
        }
        for (size_t j = 0; j < size; j++) {
            outfile<<(double)(M[i][j])<<"    ";
        }// for j
        outfile<<endl;
    }// for i
    outfile.close();
    return 0;
}
TVars   **inverse_matrix(TVars **M, size_t size) {
    int err = 0;
    size_t *POR = NULL;                                        // ������ ��� ����� ������������ �����
    POR = new size_t[size];
    if (!POR) return NULL;
    size_t PR;                                                 // ���������� ��� ����� ������������ �����
    for (size_t i = 0; i < size; i++) {
        POR[i]=i;
    }
    TVars b;
    TVars **M_inv = NULL;                                   // �������� �������
    M_inv = new TVars*[size];
    {    
        size_t i;
        for(i = 0; i < size; ++i) {
            M_inv[i] = NULL;
            M_inv[i]=new TVars[size];
            if (!M_inv[i]) break;
        }
        if (i != size) {
            while (i != 0) {
                delete[] M_inv[i--];
            }
            delete [] M_inv;
            delete[] POR;
            return NULL;
        }
    }
    for (size_t i = 0; i < size; i++) {
        for (size_t j = 0; j < size; j++) {
            if (i != j) {
                M_inv[i][j]=0;
            }
            else {
                M_inv[i][j]=1;
            }
        }
    }
    if (fabs(M[0][0])<DELT) {
        TVars mx=fabs(M[0][0]);
        size_t num=0;
        for (size_t i = 1; i < size; i++) {
            if (fabs(M[i][0])>mx) {
                mx=fabs(M[i][0]);
                num=i;
            }
        }//i
        if (num!=0) {
            err = move_line(M,size,0,num);
            if (err) {
                move_all_back(M, size, POR);
                delete[] POR;
                clear_memory(M_inv, size);
                return NULL;
            }
            err = move_line(M_inv,size,0,num);
            PR=POR[0];
            POR[0]=POR[num];
            POR[num]=PR;
            if (err) {
                move_all_back(M, size, POR);
                delete[] POR;
                clear_memory(M_inv, size);
                return NULL;
            }
        }
    }//if
    for (size_t k = 0; k < size-1; k++) {
        if (fabs(M[k][k])<DELT) {
            move_all_back(M, size, POR);
            delete[] POR;
            clear_memory(M_inv, size);
            return NULL;
        }//if
        TVars mx=fabs(M[k+1][k+1]);
        size_t line=k+1;
        for (size_t i = k+1; i < size; i++) {               // ����� �������� ��������
            if (fabs(M[i][k+1])>mx) {
                mx=fabs(M[i][k+1]);
                line=i;
            }//if
        }//i
        if (mx<DELT) {
            move_all_back(M, size, POR);
            delete[] POR;
            clear_memory(M_inv, size);
            return NULL;
        }
        err = move_line(M,size,k+1,line);
        if (err) {
            move_all_back(M, size, POR);
            delete[] POR;
            clear_memory(M_inv, size);
            return NULL;
        }
        err = move_line(M_inv,size,k+1,line);                      // ������������ �����
        PR=POR[k+1];
        POR[k+1]=POR[line];
        POR[line]=PR;
        if (err) {
            move_all_back(M, size, POR);
            delete[] POR;
            clear_memory(M_inv, size);
            return NULL;
        }
        for (size_t i = 0; i < size; i++) {
            if (i!=k) {
                TVars c=M[i][k]/M[k][k];
                for (size_t j = 0; j < size; j++) {
                    b=M[i][j]-c*(M[k][j]);                  // �������������� �������
                    M[i][j]=b;
                    b=M_inv[i][j]-c*(M_inv[k][j]);          // �������������� �������
                    M_inv[i][j]=b;
                }//j
            }//if
        }//i
    }//k
    if (fabs(M[size-1][size-1])<DELT) {
        move_all_back(M, size, POR);
        delete[] POR;
        clear_memory(M_inv, size);
        return NULL;
    }
    for (size_t i = 0; i < size-1; ++i) {
        TVars c=M[i][size-1]/M[size-1][size-1];
    //		   b=M[i][size-1]-c*(M[size-1][size-1]);        // �������������� �������
    //		   M[i][size-1]=b;
        for (size_t j = 0; j < size; j++) {
            b=M_inv[i][j]-c*(M_inv[size-1][j]);                 // �������������� �������
            M_inv[i][j]=b;
        }// j
    }// i
    for (size_t i = 0; i < size; i++) {
        for (size_t j = 0; j < size; j++) {
            M_inv[i][j]=M_inv[i][j]/M[i][i];
        }// j
    }// i
    move_all_back(M, size, POR);
    delete[] POR;
    return M_inv;
}
int     move_line(TVars **M, size_t s, size_t st, size_t fin) {
    TVars *Ln = NULL;
    Ln=new TVars[s];
    if (! Ln) return 1;
    for (size_t i = 0; i < s ; i++) {
        Ln[i]=M[st][i];
    }
    for (size_t i = 0; i < s ; i++) {
        M[st][i]=M[fin][i];
    }
    for (size_t i = 0; i < s ; i++) {
        M[fin][i]=Ln[i];
    }
    delete[] Ln;
    return 0;
}
int     move_all_back(TVars **M, size_t size, size_t *mov) {
    if (M == NULL || mov == NULL) return 1;
    int err = 0;
    int cnt = 0;
    for (size_t i = 0; i < size; ++i) {
        if (mov[i] != i) {
            err = move_line(M, size, i, mov[i]);
            if (err || cnt < 10) {
                --i;
                ++cnt;
                continue;
            }
            cnt = 0;
            if (err) return 1;
            mov[mov[i]] = mov[i];
            mov[i] = i;
        }
    }
    return 0;
}
void    clear_memory(TVars **M, size_t s) {
    if (M != NULL) {
        for (size_t i = 0; i < s; ++i) {
            if (M[i] != NULL) {
                delete[] M[i];
            }
        }
        delete[] M;
    }
}
int     incr_vort_quont(Vortex *&p_host, Vortex *&p_dev, PVortex *&v_host, PVortex *&v_dev, TVars *&d_dev, size_t &size)
{
    using namespace std;
    hipError_t cuerr;
    if (p_host != NULL && p_dev != NULL && v_host != NULL && v_dev != NULL && d_dev != NULL)
    {
        Vortex *p_dev_new = NULL;
        size_t size_n = size + INCR_STEP;
        cuerr=hipMalloc( (void**)&p_dev_new , size_n * sizeof(Vortex));
        if (cuerr != hipSuccess) {
            cout << hipGetErrorString(cuerr) << '\n';
            return 1;
        }
        cuerr = hipMemcpy (p_dev_new, p_dev, size  * sizeof(Vortex), hipMemcpyDeviceToDevice); 
        if (cuerr != hipSuccess) {
            cout << hipGetErrorString(cuerr) << '\n';
            return 1;
        }
        size += INCR_STEP;
        delete[] p_host;
        p_host = new Vortex[size];
        delete[] v_host;
        v_host = new PVortex[size];
        hipFree(p_dev);
        hipFree(d_dev);
        hipFree(v_dev);
        cuerr=hipMalloc( (void**)&d_dev, size * sizeof(TVars));
        if (cuerr != hipSuccess) {
            cout << hipGetErrorString(cuerr) << '\n';
            return 1;
        }
        cuerr=hipMalloc( (void**)&v_dev, size  * sizeof(PVortex));
        if (cuerr != hipSuccess) {
            cout << hipGetErrorString(cuerr) << '\n';
            return 1;
        }
        p_dev = p_dev_new;
        hipDeviceSynchronize();
    }
    else {
        size = INCR_STEP;
        p_host=new Vortex[size];
        v_host=new PVortex[size];
        cuerr=hipMalloc((void**)&p_dev , size * sizeof(Vortex));
        if (cuerr != hipSuccess) {
            cout << hipGetErrorString(cuerr) << '\n';
            return 1;
        }
        cuerr=hipMalloc((void**)&d_dev , size * sizeof(TVars));
        if (cuerr != hipSuccess) {
            cout << hipGetErrorString(cuerr) << '\n';
            return 1;
        }
        cuerr=hipMalloc((void**)&v_dev , size  * sizeof(PVortex));
        if (cuerr != hipSuccess) {
            cout << hipGetErrorString(cuerr) << '\n';
            return 1;
        }
    }
    srand((unsigned int)time(NULL));
    float *rnd_dev = NULL, *rnd_host = NULL;
    rnd_host = new float[INCR_STEP];
    for (int i = 0; i < INCR_STEP; ++i) {
        rnd_host[i] = (float)rand(); 
    }
    cuerr = hipMalloc((void**)&rnd_dev, INCR_STEP * sizeof(float));
    if (cuerr != hipSuccess) {
        cout << hipGetErrorString(cuerr) << '\n';
        return 1;
    }
    cuerr = hipMemcpy(rnd_dev, rnd_host, INCR_STEP * sizeof(float), hipMemcpyHostToDevice);
    if (cuerr != hipSuccess) {
        cout << hipGetErrorString(cuerr) << '\n';
        return 1;
    }
    dim3 threads = dim3(BLOCK_SIZE);
    dim3 blocks  = dim3(INCR_STEP/BLOCK_SIZE);
    // generate random numbers
    zero_Kernel <<< blocks, threads >>> (rnd_dev, p_dev, (size-INCR_STEP) );
    hipDeviceSynchronize();
    //	cuerr=hipMemcpy ( p_host , p_dev , size  * sizeof(Vortex) , hipMemcpyDeviceToHost);
    //	save_to_file_size(1);
    cuerr = hipGetLastError();
    if (cuerr != hipSuccess) {
        cout << hipGetErrorString(cuerr) << '\n';
        return 1;
    }
    return 0;
}
int     vort_creation(Vortex *pos, TVctr *V_infDev, size_t n_of_birth, size_t n_of_birth_BLOCK_S,
                     size_t n, TVars * M_Dev, TVars *d_g, tPanel *panels) {
    using namespace std;
    hipError_t cuerr = hipSuccess;
    TVars *R_p = NULL;
    cuerr=hipMalloc((void**)&R_p, (n_of_birth_BLOCK_S) * sizeof(TVars));
    if (cuerr != hipSuccess) {
        cout << hipGetErrorString(cuerr) << '\n';
        return 1;
    }
	dim3 threads1 = dim3(BLOCK_SIZE);
    dim3 blocks1  = dim3(n_of_birth_BLOCK_S/BLOCK_SIZE);
    Right_part_Kernel <<< blocks1, threads1 >>> (pos, V_infDev, n, n_of_birth_BLOCK_S, R_p, panels);
	hipDeviceSynchronize();
    cuerr = hipGetLastError();
    if (cuerr != hipSuccess) {
        cout << hipGetErrorString(cuerr) << '\n';
        return 1;
    }
	birth_Kernel<<< blocks1, threads1 >>>(pos, n, n_of_birth, n_of_birth_BLOCK_S, M_Dev, d_g, R_p, panels);
	hipDeviceSynchronize();
    cuerr = hipGetLastError();
    if (cuerr != hipSuccess) {
        cout << hipGetErrorString(cuerr) << '\n';
        return 1;
    }
    hipFree(R_p);
    return 0;
}
void start_timer(hipEvent_t &start, hipEvent_t &stop) {
    hipEventCreate(&start);	
	hipEventCreate(&stop);
	hipEventRecord(start,0);	
	hipEventSynchronize(start);
}
float stop_timer(hipEvent_t start, hipEvent_t stop) {
    hipEventRecord(stop,0);	
	hipEventSynchronize(stop);
	float time;
	hipEventElapsedTime(&time,start,stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	time=time/1000;
    return time;
}
int Speed(Vortex *pos, TVctr *v_inf, size_t s, PVortex *v, TVars *d, TVars nu, tPanel *panels) {
//    extern int current_step;
//    extern size_t n;
    hipError_t cuerr = hipSuccess;
	hipDeviceSynchronize();
	dim3 threads = dim3(BLOCK_SIZE);
    dim3 blocks  = dim3(s/BLOCK_SIZE);
//    PVortex * VEL = new PVortex[s];
//    PVortex * VELLL = new PVortex[s];
	shared_Kernel <<< blocks, threads >>> (pos, v_inf, s, v, d);
//	simple_Kernel <<< blocks, threads >>> (pos, v_inf, *n, v);
    hipDeviceSynchronize();
//    Vortex *POS = new Vortex[s];
//    cuerr=hipMemcpy (POS  , pos , s  * sizeof(Vortex) , hipMemcpyDeviceToHost);
//    cuerr=hipMemcpy (VEL  , v , s  * sizeof(PVortex) , hipMemcpyDeviceToHost);
//    save_vel_to_file(POS, VEL, n, current_step, 0);
    cuerr=hipGetLastError(); 
	if (cuerr != hipSuccess) {               
		std::cout <<hipGetErrorString(cuerr);
		return 1;            
	}//if

//	TVars* dd=new TVars[s];
//    hipMemcpy(dd,d,s * sizeof(TVars),hipMemcpyDeviceToHost);
//    save_d(dd, s, current_step);
//    delete[]dd;

	diffusion_Kernel <<< blocks, threads >>> (pos, s, v, d, nu);
//	cuerr=hipMemcpy (POS  , posDev , size  * sizeof(Vortex) , hipMemcpyDeviceToHost);
//	save_to_file(j);
	hipDeviceSynchronize();
//	cuerr=hipMemcpy (VELLL  , v , s  * sizeof(PVortex) , hipMemcpyDeviceToHost);
/*
    for (size_t sss = 0; sss < s; ++sss) {
        VEL[sss].v[0] = VELLL[sss].v[0] - VEL[sss].v[0];
        VEL[sss].v[1] = VELLL[sss].v[1] - VEL[sss].v[1];
    }
	save_vel_to_file(POS, VEL, n, current_step, 1);
*/
    cuerr=hipGetLastError(); 
	if (cuerr != hipSuccess) {               
		std::cout <<hipGetErrorString(cuerr);
		return 1;            
	}//if
	diffusion_2_Kernel <<< blocks, threads >>> (pos, s, v, d, nu, panels);
//	cuerr=hipMemcpy (VEL  , VDev , size  * sizeof(PVortex) , hipMemcpyDeviceToHost);
//	stf(j,1);
	hipDeviceSynchronize();
/*
    cuerr=hipMemcpy (VEL  , v , s  * sizeof(PVortex) , hipMemcpyDeviceToHost);
    for (size_t sss = 0; sss < s; ++sss) {
        VELLL[sss].v[0] = VEL[sss].v[0] - VELLL[sss].v[0];
        VELLL[sss].v[1] = VEL[sss].v[1] - VELLL[sss].v[1];
    }
    save_vel_to_file(POS, VELLL, n, current_step, 2);
    save_vel_to_file(POS, VEL, n, current_step, 3);
*/
/*	
	TVars *dd=new TVars[size];
    hipMemcpy(dd,d,size * sizeof(TVars),hipMemcpyDeviceToHost);
    cout<<"d= "<<dd[0]<<endl;
    delete[]dd;
	TVars *ddt=new TVars;
	cuerr=hipMemcpy (ddt  , den ,  sizeof(TVars) , hipMemcpyDeviceToHost);
	cout<<"nu*y=  "<<(*ddt)<<endl;
	TVctr V_inf;
	cuerr=hipMemcpy (VEL  , VDev , size  * sizeof(PVortex) , hipMemcpyDeviceToHost);
	cuerr=hipMemcpy (V_inf  , V_infDev , sizeof(TVctr) , hipMemcpyDeviceToHost);
	cout<<"V=  "<<VEL[0].v[0]<<endl;
*/
//	hipDeviceSynchronize();
//    cuerr=hipMemcpy2D ( M3 , nb , cDev, pitch , nb , &n , hipMemcpyDeviceToHost);   
   	cuerr=hipGetLastError(); 
	if (cuerr != hipSuccess) {               
		std::cout <<hipGetErrorString(cuerr);
		return 1;            
	}//if
	return 0;
}

void save_vel_to_file(Vortex *POS, PVortex *VEL, size_t size, int _step, int stage) {
    using namespace std;
    char *fname1;
    fname1 = "velocities/Vel";
    char *fname2;
    fname2 = ".txt";
    char *fzero;
    fzero = "0";
    char fstep[8];
    char fname[20];
    fname[0] = '\0';
    char stage_str[5];
    itoa(stage, stage_str, 10);
    itoa(_step,fstep,10);
    strcat(fname,fname1);
    strcat(fname, stage_str);
    if (_step<10) strcat(fname,fzero);
    if (_step<100) strcat(fname,fzero);
    if (_step<1000) strcat(fname,fzero);
    if (_step<10000) strcat(fname,fzero);
    //	if (_step<100000) strcat(fname,fzero);
    strcat(fname,fstep);
    strcat(fname,fname2);
    ofstream outfile;
    outfile.open(fname);
    // ���������� ����� ������ � ������
    outfile << (size) << endl;
    for (size_t i = 0; i < (size); ++i) {
        outfile<<(int)(i)<<" "<<(double)(POS[i].r[0])<<" "<<(double)(POS[i].r[1])<<" "<<(double)(VEL[i].v[0])<<" "<<(double)(VEL[i].v[1])<<endl;
        //      outfile<<(double)(d[i])<<" "<<(double)(POS[i].r[0])<<" "<<(double)(POS[i].r[1])<<" "<<(double)(POS[i].g)<<endl;     
    }//for i
    outfile.close();
} //save_to_file

void save_d(double *d, size_t size, int _step) {
    using namespace std;
    char *fname1;
    fname1 = "ddd/d";
    char *fname2;
    fname2 = ".txt";
    char *fzero;
    fzero = "0";
    char fstep[8];
    char fname[20];
    fname[0] = '\0';
    itoa(_step,fstep,10);
    strcat(fname,fname1);
    if (_step<10) strcat(fname,fzero);
    if (_step<100) strcat(fname,fzero);
    if (_step<1000) strcat(fname,fzero);
    if (_step<10000) strcat(fname,fzero);
    //	if (_step<100000) strcat(fname,fzero);
    strcat(fname,fstep);
    strcat(fname,fname2);
    ofstream outfile;
    outfile.open(fname);
    // ���������� ����� ������ � ������
    outfile << (size) << endl;
    for (size_t i = 0; i < (size); ++i) {
        outfile<<(int)(i)<<" "<<d[i]<<endl;
        //      outfile<<(double)(d[i])<<" "<<(double)(POS[i].r[0])<<" "<<(double)(POS[i].r[1])<<" "<<(double)(POS[i].g)<<endl;     
    }//for i
    outfile.close();
} //save_to_file

int Step(Vortex *pos, PVortex *V, size_t &n, size_t s, TVars *d_g, PVortex *F_p, TVars *M, tPanel *panels) {
	hipError_t cuerr = hipSuccess;
	TVars *d_g_Dev = NULL;
	cuerr=hipMalloc((void**)&d_g_Dev, n * sizeof(TVars)); 
	if (cuerr!= hipSuccess) {               
		std::cout <<hipGetErrorString(cuerr) << '\n';
		return 1;            
	}//if
    PVortex *F_p_dev = NULL;
    TVars *M_dev = NULL;
    cuerr=hipMalloc((void**)&F_p_dev, n * sizeof(PVortex)); 
	if (cuerr!= hipSuccess) {               
		std::cout <<hipGetErrorString(cuerr) << '\n';
		return 1;            
	}//if
    cuerr=hipMalloc((void**)&M_dev, n * sizeof(TVars)); 
	if (cuerr!= hipSuccess) {               
		std::cout <<hipGetErrorString(cuerr) << '\n';
		return 1;            
	}//if
//	TVars d_g_h;
//	cuerr=hipMemcpy ( &d_g_h, d_g , sizeof(TVars) , hipMemcpyDeviceToHost);
//  std::cout << "D_g_before = " << d_g_h << '\n';
    dim3 threads = dim3(BLOCK_SIZE);
    dim3 blocks  = dim3(s/BLOCK_SIZE);
	step_Kernel <<< blocks, threads >>> (pos, V, d_g_Dev, F_p_dev, M_dev, n, panels);
    hipDeviceSynchronize();
    cuerr=hipGetLastError(); 
	if (cuerr!= hipSuccess) {               
		std::cout <<hipGetErrorString(cuerr) << '\n';
		return 1;            
	}//if

//	cuerr=hipMemcpy ( POS , posDev , size  * sizeof(Vortex) , hipMemcpyDeviceToHost);
//	save_to_file_size((*n)+1);

	summ_Kernel <<< dim3(1),dim3(1) >>> (d_g_Dev, d_g, F_p_dev, F_p, M_dev, M, n);
	hipDeviceSynchronize();
    cuerr=hipGetLastError(); 
	if (cuerr!= hipSuccess) {               
		std::cout <<hipGetErrorString(cuerr) << '\n';
		return 1;            
	}//if
	hipFree(d_g_Dev);
    hipFree(F_p_dev);
    hipFree(M_dev);
	TVars d_g_h = 0.0;
	cuerr=hipMemcpy ( &d_g_h, d_g , sizeof(TVars) , hipMemcpyDeviceToHost);
//	std::cout << "d_g = " << d_g_h << '\n';

	size_t *n_dev = NULL;
	cuerr = hipMalloc( (void**)&n_dev ,  sizeof(size_t)); 
	if (cuerr!= hipSuccess) {               
		std::cout <<hipGetErrorString(cuerr) << '\n';
		return 1;            
	}//if
	cuerr = hipMemcpy(n_dev, &n, sizeof(size_t), hipMemcpyHostToDevice);
	if (cuerr!= hipSuccess) {               
		std::cout <<hipGetErrorString(cuerr) << '\n';
		return 1;            
	}//if
	sort_Kernel <<< dim3(1), dim3(1) >>> (pos,n_dev);
    hipDeviceSynchronize();
    cuerr=hipGetLastError(); 
	if (cuerr!= hipSuccess) {               
		std::cout <<hipGetErrorString(cuerr) << '\n';
		return 1;            
	}//if
	cuerr = hipMemcpy(&n,n_dev,sizeof(size_t), hipMemcpyDeviceToHost);
	if (cuerr!= hipSuccess) {               
		std::cout <<hipGetErrorString(cuerr) << '\n';
		return 1;            
	}//if
	hipFree(n_dev);
//    std::cout << "first collapse\n";
	for (int cc = 0; cc < NCOL; ++cc) {
		int *Setx = NULL;
		int *Sety = NULL;
		int *COL = NULL;
		cuerr=hipMalloc (&Setx, n * sizeof( int ));
		cuerr=hipMalloc (&Sety, n * sizeof( int ));
		cuerr=hipMalloc (&COL, n * sizeof( int ));
		
		first_setka_Kernel <<< blocks, threads >>> (pos, n, Setx, Sety, COL);
		hipFree(Setx);
		hipFree(Sety);
//		int *COLD;
//		COLD= new int [n];
//		hipMemcpy(COLD, COL, n * sizeof(int), hipMemcpyDeviceToHost);
/*
        int sss = 0;
		for(int gg = 0; gg < n; gg++) {
			if (COLD[gg] >= 0) sss += 1;
		}
		std::cout << cc << ' ' << sss << '\n';
		if (sss==0) cc=10;
		delete[] COLD;
*/
        hipDeviceSynchronize();
		first_collapse_Kernel <<< dim3(1), dim3(1) >>> (pos, COL, n);
		hipFree(COL);
		hipMalloc( (void**)&n_dev ,  sizeof(size_t));
		hipMemcpy(n_dev, &n, sizeof(size_t), hipMemcpyHostToDevice);
		hipDeviceSynchronize();
		sort_Kernel <<< dim3(1), dim3(1) >>> (pos, n_dev);
		hipDeviceSynchronize();
		hipMemcpy(&n, n_dev, sizeof(size_t), hipMemcpyDeviceToHost);
		hipFree(n_dev);
	}
//    std::cout << "second collapse\n";
    for (int cc = 0; cc < NCOL; ++cc) {
        int *Setx = NULL;
        int *Sety = NULL;
        int *COL = NULL;
        cuerr=hipMalloc (&Setx, n * sizeof( int ));
        cuerr=hipMalloc (&Sety, n * sizeof( int ));
        cuerr=hipMalloc (&COL, n * sizeof( int ));

        second_setka_Kernel <<< blocks, threads >>> (pos, n, Setx, Sety, COL);
        hipFree(Setx);
        hipFree(Sety);
/*
        int *COLD;
        COLD= new int [n];
        hipMemcpy(COLD, COL, n * sizeof(int), hipMemcpyDeviceToHost);
        int sss = 0;
        for(int gg = 0; gg < n; gg++) {
            if (COLD[gg] >= 0) sss += 1;
        }
        std::cout << cc << ' ' << sss << '\n';
        if (sss==0) cc=10;
        delete[] COLD;
*/
        hipDeviceSynchronize();
        second_collapse_Kernel <<< dim3(1), dim3(1) >>> (pos, COL, n);
        hipFree(COL);
        hipMalloc( (void**)&n_dev ,  sizeof(size_t));
        hipMemcpy(n_dev, &n, sizeof(size_t), hipMemcpyHostToDevice);
        hipDeviceSynchronize();
        sort_Kernel <<< dim3(1), dim3(1) >>> (pos, n_dev);
        hipDeviceSynchronize();
        hipMemcpy(&n, n_dev, sizeof(size_t), hipMemcpyDeviceToHost);
        hipFree(n_dev);
    }
	return 0;
}



int velocity_control(Vortex *pos, TVctr *V_inf, int n, PVortex *Contr_points, PVortex *V, int *n_v) {
    hipError_t cuerr;
    dim3 threads = dim3(50);
    dim3 blocks  = dim3(10);
    velocity_control_Kernel <<< blocks, threads >>> (pos, V_inf, n, Contr_points, V, n_v);
    hipDeviceSynchronize();    
    cuerr = hipGetLastError();    
    if (cuerr != hipSuccess) {        
        std::cout << hipGetErrorString(cuerr) << '\n';
        return 1;
    }
    return 0;
}